#include "bertCommon.h"
#include "trt_tensor.hpp"

#include <hipDNN.h>
#include <iostream>
#include <ctime>
#include <cstdlib>
#include <cassert>


#define checkCUDNN(expression)                               \
{                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
        std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
        std::exit(EXIT_FAILURE);                               \
    }                                                        \
}

int test_swish() {
    ::srand(::time(0));
    std::cout << ::rand() << std::endl;

    constexpr int batch_size = 4;
    constexpr int channel_in = 3;
    constexpr int height_in = 112;
    constexpr int width_in = 112;

    constexpr int channel_out = channel_in;
    constexpr int height_out = 112;
    constexpr int width_out = 112;

    TRT::Tensor input_tensor(std::vector<int>{batch_size, channel_in, height_in, width_in});
    TRT::Tensor out_tensor(std::vector<int>{batch_size, channel_in, height_out, width_out});
    
    const float alpha1 = 1;
    const float alpha2 = 0;

    auto input_ptr_cpu = input_tensor.cpu<float>();

    for(int i = 0; i < input_tensor.numel(); ++i) 
    {
        input_ptr_cpu[i] = float(rand() % 100000) / 100000;
    }
    input_tensor.save_to_file("input_tensor.npz");

    hipStream_t stream = out_tensor.get_stream();
    
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));
    checkCUDNN(hipdnnSetStream(cudnn, stream));

    hipdnnActivationDescriptor_t activation_descriptor;
    checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
    checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
                                            /*mode=*/HIPDNN_ACTIVATION_SWISH,
                                            /*reluNanOpt=*/HIPDNN_PROPAGATE_NAN,
                                            /*relu_coef=*/0));
    // checkCUDNN(cudnnSetActivationDescriptorSwishBeta(activation_descriptor, 0));

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                          /*format=*/HIPDNN_TENSOR_NCHW,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/batch_size,
                                          /*channels=*/channel_in,
                                          /*image_height=*/height_in,
                                          /*image_width=*/width_in));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                      /*format=*/HIPDNN_TENSOR_NCHW,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/batch_size,
                                      /*channels=*/channel_out,
                                      /*image_height=*/height_out,
                                      /*image_width=*/width_out));

    auto input_tensor_gpu = input_tensor.to_gpu(true).gpu<float>();
    auto outptr_gpu = out_tensor.to_gpu().gpu<float>();

    checkCUDNN(hipdnnActivationForward(
        cudnn,
        activation_descriptor,
        &alpha1,
        input_descriptor,
        input_tensor_gpu,
        &alpha2,
        output_descriptor,
        outptr_gpu));

    out_tensor.to_cpu(true);
    out_tensor.save_to_file("out_tensor.npz");

    return 0;
}

int main() {

    test_swish();

    return 0;
}